#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <csignal>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <iterator>
#include <sstream>
#include <string>
#include <vector>

#include "hip/hip_runtime.h"
#include "mpi.h"

#define FATAL(description)                                      \
    do {                                                        \
        std::cerr << "Error in " << __FILE__ << ":" << __LINE__ \
                  << ". Message: " << description << std::endl; \
        exit(0);                                                \
    } while (0)

#define CHECK_CUDART(call)                  \
    do {                                    \
        hipError_t res = call;             \
        if (res != hipSuccess) {           \
            FATAL(hipGetErrorString(res)); \
        }                                   \
    } while (0)

#define CHECK_MPI(call)                        \
    do {                                       \
        int res = call;                        \
        if (res != MPI_SUCCESS) {              \
            char desc[MPI_MAX_ERROR_STRING];   \
            int len;                           \
            MPI_Error_string(res, desc, &len); \
            MPI_Finalize();                    \
            FATAL(desc);                       \
        }                                      \
    } while (0)

void handle_signals() {
    auto nop_handler = [](int sig) {
        std::cerr << "Unexpected signal received: " << sig << std::endl;
        exit(0);
    };
    std::signal(SIGSEGV, nop_handler);
    std::signal(SIGABRT, nop_handler);
}

struct MPIContext {
    MPIContext(int *argc, char ***argv) { CHECK_MPI(MPI_Init(argc, argv)); }
    ~MPIContext() {
        CHECK_MPI(MPI_Barrier(MPI_COMM_WORLD));
        CHECK_MPI(MPI_Finalize());
    }
};

template <typename T>
struct Vector3 {
    T x, y, z;

    friend std::istream &operator>>(std::istream &is, Vector3 &v) {
        is >> v.x >> v.y >> v.z;
        return is;
    }

    friend std::ostream &operator<<(std::ostream &os, const Vector3 &v) {
        os << '[' << v.x << ' ' << v.y << ' ' << v.z << ']';
        return os;
    }
};

using Vector3d = Vector3<double>;

struct CylindricalMovementParams {
    double r0, z0, phi0, ar, az, wr, wz, wphi, pr, pz;

    friend std::istream &operator>>(std::istream &is,
                                    CylindricalMovementParams &p) {
        is >> p.r0 >> p.z0 >> p.phi0 >> p.ar >> p.az >> p.wr >> p.wz >>
            p.wphi >> p.pr >> p.pz;
        return is;
    }

    friend std::ostream &operator<<(std::ostream &os,
                                    const CylindricalMovementParams &p) {
        os << p.r0 << ' ' << p.z0 << ' ' << p.phi0 << ' ' << p.ar << ' ' << p.az
           << ' ' << p.wr << ' ' << p.wz << ' ' << p.wphi << ' ' << p.pr << ' '
           << p.pz;
        return os;
    }
};

struct FigureParams {
    Vector3d center;
    Vector3d color;
    double radius;
    double kreflection, krefraction;
    int nlights;

    friend std::istream &operator>>(std::istream &is, FigureParams &p) {
        is >> p.center >> p.color >> p.radius >> p.kreflection >>
            p.krefraction >> p.nlights;
        return is;
    }

    friend std::ostream &operator<<(std::ostream &os, const FigureParams &p) {
        os << p.center << ' ' << p.color << ' ' << p.radius << ' '
           << p.kreflection << ' ' << p.krefraction << ' ' << p.nlights;
        return os;
    }
};

struct FloorParams {
    Vector3d a, b, c, d;
    std::string texture_path;
    Vector3d color;
    double kreflection;

    friend std::istream &operator>>(std::istream &is, FloorParams &p) {
        is >> p.a >> p.b >> p.c >> p.d >> p.texture_path >> p.color >>
            p.kreflection;
        return is;
    }

    friend std::ostream &operator<<(std::ostream &os, const FloorParams &p) {
        os << p.a << ' ' << p.b << ' ' << p.c << ' ' << p.d << ' '
           << p.texture_path << ' ' << p.color << ' ' << p.kreflection;
        return os;
    }
};

struct LightParams {
    Vector3d pos;
    Vector3d color;

    friend std::istream &operator>>(std::istream &is, LightParams &p) {
        is >> p.pos >> p.color;
        return is;
    }

    friend std::ostream &operator<<(std::ostream &os, const LightParams &p) {
        os << p.pos << ' ' << p.color;
        return os;
    }
};

struct Params {
    int nframes;
    std::string output_pattern;
    int w, h;
    double angle;
    CylindricalMovementParams camera_center, camera_dir;
    FigureParams hex, octa, icos;
    FloorParams floor;
    int nlights;
    std::vector<LightParams> lights;

    friend std::istream &operator>>(std::istream &is, Params &p) {
        is >> p.nframes >> p.output_pattern >> p.w >> p.h >> p.angle >>
            p.camera_center >> p.camera_dir >> p.hex >> p.octa >> p.icos >>
            p.floor >> p.nlights;
        p.lights.resize(p.nlights);
        for (auto &it : p.lights) is >> it;
        return is;
    }

    friend std::ostream &operator<<(std::ostream &os, const Params &p) {
        os << p.nframes << ' ' << p.output_pattern << '\n'
           << p.w << ' ' << p.h << ' ' << p.angle << '\n'
           << "camera center: " << p.camera_center << '\n'
           << "camera dir: " << p.camera_dir << '\n'
           << "hex: " << p.hex << '\n'
           << "octa: " << p.octa << '\n'
           << "icos: " << p.icos << '\n'
           << "floor: " << p.floor << '\n'
           << "nlights: " << p.nlights << '\n';
        for (auto &it : p.lights) os << it;
        return os;
    }
};

struct Trig {
    Vector3d a;
    Vector3d b;
    Vector3d c;
    Vector3d color;
};

static std::vector<Trig> scene_trigs;

std::vector<std::string> split_string(const std::string &s, char d) {
    std::vector<std::string> result;
    std::stringstream ss(s);
    std::string word;
    while (getline(ss, word, d)) {
        result.push_back(word);
    }
    return result;
}

double dot_product(Vector3d a, Vector3d b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

Vector3d cross_product(Vector3d a, Vector3d b) {
    return {a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z,
            a.x * b.y - a.y * b.x};
}

double norm(Vector3d v) { return sqrt(dot_product(v, v)); }

Vector3d normalize(Vector3d v) {
    double l = norm(v);
    return {v.x / l, v.y / l, v.z / l};
}

Vector3d diff(Vector3d a, Vector3d b) {
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

Vector3d add(Vector3d a, Vector3d b) {
    return {a.x + b.x, a.y + b.y, a.z + b.z};
}

Vector3d mult(Vector3d a, Vector3d b, Vector3d c, Vector3d v) {
    return {a.x * v.x + b.x * v.y + c.x * v.z,
            a.y * v.x + b.y * v.y + c.y * v.z,
            a.z * v.x + b.z * v.y + c.z * v.z};
}

Vector3d mult(Vector3d a, double k) { return {k * a.x, k * a.y, k * a.z}; }

Vector3d div(Vector3d a, double k) { return {a.x / k, a.y / k, a.z / k}; }

uchar4 color_from_normalized(Vector3d v) {
    return make_uchar4(255. * v.x, 255. * v.y, 255. * v.z, 0u);
}

void import_obj_file(const std::string &filepath, const FigureParams &fp) {
    std::ifstream is(filepath);
    if (!is) {
        std::string desc = "can't open " + filepath;
        FATAL(desc);
    }

    double r = 0;
    std::vector<Vector3d> vertices;
    std::vector<Trig> figure_trigs;
    std::string line;
    while (std::getline(is, line)) {
        std::vector<std::string> buffer = split_string(line, ' ');
        if (line.empty()) {
            continue;
        } else if (buffer[0] == "v") {
            double x = std::stod(buffer[2]);
            double y = std::stod(buffer[3]);
            double z = std::stod(buffer[4]);

            vertices.push_back({x, y, z});
        } else if (buffer[0] == "f") {
            std::vector<std::string> indexes = split_string(buffer[1], '/');
            Vector3d a = vertices[std::stoi(indexes[0]) - 1];
            indexes = split_string(buffer[2], '/');
            Vector3d b = vertices[std::stoi(indexes[0]) - 1];
            indexes = split_string(buffer[3], '/');
            Vector3d c = vertices[std::stoi(indexes[0]) - 1];

            r = std::max(r, norm(a));
            r = std::max(r, norm(b));
            r = std::max(r, norm(c));

            figure_trigs.push_back(Trig{a, b, c, fp.color});
        }
    }
    for (auto &it : figure_trigs) {
        double k = fp.radius / r;
        Vector3d a = add(mult(it.a, k), fp.center);
        Vector3d b = add(mult(it.b, k), fp.center);
        Vector3d c = add(mult(it.c, k), fp.center);
        scene_trigs.push_back({a, b, c, it.color});
    }
}

void add_floor(const FloorParams &fp) {
    scene_trigs.push_back({fp.a, fp.b, fp.c, fp.color});
    scene_trigs.push_back({fp.a, fp.d, fp.c, fp.color});
}

uchar4 ray(Vector3d pos, Vector3d dir) {
    int k, k_min = -1;
    double ts_min;
    for (k = 0; k < scene_trigs.size(); k++) {
        Vector3d e1 = diff(scene_trigs[k].b, scene_trigs[k].a);
        Vector3d e2 = diff(scene_trigs[k].c, scene_trigs[k].a);
        Vector3d p = cross_product(dir, e2);
        double div = dot_product(p, e1);
        if (fabs(div) < 1e-10) continue;
        Vector3d t = diff(pos, scene_trigs[k].a);
        double u = dot_product(p, t) / div;
        if (u < 0.0 || u > 1.0) continue;
        Vector3d q = cross_product(t, e1);
        double v = dot_product(q, dir) / div;
        if (v < 0.0 || v + u > 1.0) continue;
        double ts = dot_product(q, e2) / div;
        if (ts < 0.0) continue;
        if (k_min == -1 || ts < ts_min) {
            k_min = k;
            ts_min = ts;
        }
    }
    if (k_min == -1) return {0, 0, 0, 0};
    return color_from_normalized(scene_trigs[k_min].color);
}

void render(Vector3d pc, Vector3d pv, int w, int h, double angle,
            uchar4 *data) {
    int i, j;
    double dw = 2.0 / (w - 1.0);
    double dh = 2.0 / (h - 1.0);
    double z = 1.0 / tan(angle * M_PI / 360.0);
    Vector3d bz = normalize(diff(pv, pc));
    Vector3d bx = normalize(cross_product(bz, {0.0, 0.0, 1.0}));
    Vector3d by = normalize(cross_product(bx, bz));
    for (i = 0; i < w; i++)
        for (j = 0; j < h; j++) {
            Vector3d v = {-1.0 + dw * i, (-1.0 + dh * j) * h / w, z};
            Vector3d dir = mult(bx, by, bz, v);
            data[(h - 1 - j) * w + i] = ray(pc, normalize(dir));
        }
}

int main(int argc, char *argv[]) {
    handle_signals();
    // MPIContext ctx(&argc, &argv);

    Params params;
    std::cin >> params;
    std::cerr << params << std::endl;

    import_obj_file("hex.obj", params.hex);
    add_floor(params.floor);

    int w = 640, h = 480;
    uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    Vector3d pc, pv;

    // build_space();

    for (int frame = 0; frame < params.nframes; ++frame) {
        pc = (Vector3d){6.0 * sin(0.05 * frame), 6.0 * cos(0.05 * frame),
                        5.0 + 2.0 * sin(0.1 * frame)};
        pv = (Vector3d){3.0 * sin(0.05 * frame + M_PI),
                        3.0 * cos(0.05 * frame + M_PI), 0.0};
        render(pc, pv, w, h, 120.0, data);

        char output_path[256];
        sprintf(output_path, params.output_pattern.data(), frame);
        printf("%d: %s\n", frame, output_path);

        FILE *out = fopen(output_path, "wb");
        fwrite(&w, sizeof(int), 1, out);
        fwrite(&h, sizeof(int), 1, out);
        fwrite(data, sizeof(uchar4), w * h, out);
        fclose(out);
    }
    free(data);
    return 0;
}